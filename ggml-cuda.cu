#include "hip/hip_runtime.h"
#include <cstdint>
#include <stdint.h>
#include <stdio.h>
#include <atomic>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "ggml-cuda.h"
#include "ggml.h"

static_assert(sizeof(half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "cuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);    \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

typedef void (*to_fp32_cuda_t)(const void * x, float * y, int k, hipStream_t stream);

#define QK4_0 32
typedef struct {
    float   d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(float) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
typedef struct {
    float   d;              // delta
    float   m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(float) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK4_2 16
typedef struct {
    half  d;                // delta
    uint8_t qs[QK4_2 / 2];  // nibbles / quants
} block_q4_2;
static_assert(sizeof(block_q4_2) == sizeof(ggml_fp16_t) + QK4_2 / 2, "wrong q4_2 block size/padding");

#define QK5_0 32
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
typedef struct {
    half d;                 // delta
    half m;                 // min
    uint32_t qh;            // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
typedef struct {
    float   d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(float) + QK8_0, "wrong q8_0 block size/padding");

static __global__ void dequantize_block_q4_0(const void * vx, float * y) {
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_0 + l + 0] = v0;
        y[i*QK4_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_1(const void * vx, float * y) {
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK4_1 + l + 0] = v0;
        y[i*QK4_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_2(const void * vx, float * y) {
    const block_q4_2 * x = (const block_q4_2 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_2; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_2 + l + 0] = v0;
        y[i*QK4_2 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q5_0(const void * vx, float * y) {
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    uint32_t qh;
    memcpy(&qh, x[i].qh, sizeof(qh));

    for (int l = 0; l < QK5_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vh0 = ((qh & (1 << (l + 0))) >> (l + 0)) << 4;
        const int8_t vh1 = ((qh & (1 << (l + 1))) >> (l + 1)) << 4;

        const int8_t vi0 = ((vi & 0xf) | vh0);
        const int8_t vi1 = ((vi >>  4) | vh1);

        const float v0 = (vi0 - 16)*d;
        const float v1 = (vi1 - 16)*d;

        y[i*QK5_0 + l + 0] = v0;
        y[i*QK5_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q5_1(const void * vx, float * y) {
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    const uint32_t qh = x[i].qh;

    for (int l = 0; l < QK5_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vh0 = ((qh & (1 << (l + 0))) >> (l + 0)) << 4;
        const int8_t vh1 = ((qh & (1 << (l + 1))) >> (l + 1)) << 4;

        const int8_t vi0 = (vi & 0xf) | vh0;
        const int8_t vi1 = (vi >>  4) | vh1;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK5_1 + l + 0] = v0;
        y[i*QK5_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q8_0(const void * vx, float * y) {
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const int8_t * pp = x[i].qs;

    for (int l = 0; l < QK8_0; l++) {
        const int8_t vi = pp[l];

        y[i*QK8_0 + l] = vi*d;
    }
}

static void dequantize_row_q4_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_0;
    dequantize_block_q4_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q4_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_1;
    dequantize_block_q4_1<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q4_2_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_2;
    dequantize_block_q4_2<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_0;
    dequantize_block_q5_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_1;
    dequantize_block_q5_1<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q8_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK8_0;
    dequantize_block_q8_0<<<nb, 1, 0, stream>>>(vx, y);
}

static __global__ void convert_fp16_to_fp32(const void * vx, float * y) {
    const half * x = (const half *) vx;

    const int i = blockIdx.x;

    y[i] = __half2float(x[i]);
}

static void convert_fp16_to_fp32_cuda(const void * x, float * y, int k, hipStream_t stream) {
    convert_fp16_to_fp32<<<k, 1, 0, stream>>>(x, y);
}

static to_fp32_cuda_t ggml_get_to_fp32_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_TYPE_Q4_2:
            return dequantize_row_q4_2_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        case GGML_TYPE_F16:
            return convert_fp16_to_fp32_cuda;
        default:
            return nullptr;
    }
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 16

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

static void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}

static hipblasHandle_t g_cublasH = nullptr;
static hipStream_t g_cudaStream = nullptr;
static hipStream_t g_cudaStream2 = nullptr;
static hipEvent_t g_cudaEvent = nullptr;

void ggml_init_cublas() {
    if (g_cublasH == nullptr) {
        // create cublas handle, bind a stream
        CUBLAS_CHECK(hipblasCreate(&g_cublasH));
        CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(g_cublasH, g_cudaStream));
        // enable tensor cores
        CUBLAS_CHECK(hipblasSetMathMode(g_cublasH, HIPBLAS_TENSOR_OP_MATH));

        // create additional stream and event for synchronization
        CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStream2, hipStreamNonBlocking));
        CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvent, hipEventDisableTiming));

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, NULL));
    }
}

void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

static hipError_t ggml_cuda_h2d_tensor_2d(void * dst, const struct ggml_tensor * src, uint64_t i3, uint64_t i2, hipStream_t stream) {
    const uint64_t ne0 = src->ne[0];
    const uint64_t ne1 = src->ne[1];
    const uint64_t nb0 = src->nb[0];
    const uint64_t nb1 = src->nb[1];
    const uint64_t nb2 = src->nb[2];
    const uint64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const size_t ts = ggml_type_size(type);
    const size_t bs = ggml_blck_size(type);

    const void * x = (const void *) ((const char *) src->data + i2*nb2 + i3*nb3);
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst, x, ne1*nb1, hipMemcpyHostToDevice, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst, ts*ne0/bs, x, nb1, ts*ne0/bs, ne1, hipMemcpyHostToDevice, stream);
    } else {
        for (uint64_t i1 = 0; i1 < ne1; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) ((char *) dst + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, hipMemcpyHostToDevice, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

static void ggml_cuda_mul_mat_f32(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;

    size_t x_size, y_size, d_size;
    float * d_X = (float *) ggml_cuda_pool_malloc(sizeof(float) * x_ne, &x_size);
    float * d_Y = (float *) ggml_cuda_pool_malloc(sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(sizeof(float) * d_ne, &d_size);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            // copy data to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(d_X, src0, i03, i02, g_cudaStream));
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(d_Y, src1, i03, i02, g_cudaStream));

            // compute
            CUBLAS_CHECK(
                hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, d_X, ne00,
                                d_Y, ne10,
                        &beta,  d_D, ne01));

            // copy data to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, d_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, g_cudaStream));
        }
    }

    CUDA_CHECK(hipStreamSynchronize(g_cudaStream));
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
}

static void ggml_cuda_mul_mat_q(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];
    const ggml_type type = src0->type;

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;

    size_t x_size, y_size, d_size, q_size;
    float * d_X = (float *) ggml_cuda_pool_malloc(sizeof(float) * x_ne, &x_size);
    float * d_Y = (float *) ggml_cuda_pool_malloc(sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(sizeof(float) * d_ne, &d_size);
    void  * d_Q = (void  *) ggml_cuda_pool_malloc(ggml_type_size(type) * x_ne / ggml_blck_size(type), &q_size);

    const to_fp32_cuda_t to_fp32_cuda = ggml_get_to_fp32_cuda(type);
    GGML_ASSERT(to_fp32_cuda != NULL);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            // copy and convert to fp32 on device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(d_Q, src0, i03, i02, g_cudaStream2));

            to_fp32_cuda(d_Q, d_X, x_ne, g_cudaStream2);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventRecord(g_cudaEvent, g_cudaStream2));

            // copy data to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(d_Y, src1, i03, i02, g_cudaStream));

            // wait for conversion
            CUDA_CHECK(hipStreamWaitEvent(g_cudaStream, g_cudaEvent, 0));

            // compute
            CUBLAS_CHECK(
                hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, d_X, ne00,
                                d_Y, ne10,
                        &beta,  d_D, ne01));

            // copy data to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, d_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, g_cudaStream));
        }
    }

    CUDA_CHECK(hipStreamSynchronize(g_cudaStream));
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
    ggml_cuda_pool_free(d_Q, q_size);
}

bool ggml_cuda_can_mul_mat(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    const int64_t ne10 = src1->ne[0];

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    // TODO: find the optimal values for these
    if ((src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16 || ggml_is_quantized(src0->type)) &&
        src1->type == GGML_TYPE_F32 &&
        dst->type == GGML_TYPE_F32 &&
        (ne0 >= 32 && ne1 >= 32 && ne10 >= 32)) {

        return true;
    }

    return false;
}

void ggml_cuda_mul_mat(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(ggml_cuda_can_mul_mat(src0, src1, dst));

    const ggml_type type = src0->type;

    if (type == GGML_TYPE_F32) {
        ggml_cuda_mul_mat_f32(src0, src1, dst);
    }
    else if (type == GGML_TYPE_F16 || ggml_is_quantized(type)) {
        ggml_cuda_mul_mat_q(src0, src1, dst);
    }
    else {
        GGML_ASSERT(false);
    }
}
