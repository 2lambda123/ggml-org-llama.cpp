#include "hip/hip_runtime.h"
static const int GGML_CUDA_MAX_SUBSTREAMS = 1;
static const bool GGML_CUDA_SEQ_COMPUTE = true;

#define WARP_SIZE 32
#define CUDA_ADD_BLOCK_SIZE 256
#define CUDA_MUL_BLOCK_SIZE 256
#define CUDA_SILU_BLOCK_SIZE 256
#define CUDA_CPY_BLOCK_SIZE 32
#define CUDA_SCALE_BLOCK_SIZE 256
#define CUDA_ROPE_BLOCK_SIZE 256
#define CUDA_DIAG_MASK_INF_BLOCK_SIZE 32
#define CUDA_DEQUANTIZE_BLOCK_SIZE 256
#define CUDA_GET_ROWS_BLOCK_SIZE 256
#define CUDA_QUANTIZE_BLOCK_SIZE 256

// dmmv = dequantize_mul_mat_vec
#ifndef GGML_CUDA_DMMV_X
#define GGML_CUDA_DMMV_X 32
#endif
#ifndef GGML_CUDA_DMMV_Y
#define GGML_CUDA_DMMV_Y 1
#endif
#ifndef GGML_CUDA_MMV_Y
#define GGML_CUDA_MMV_Y 1
#endif


#ifndef K_QUANTS_PER_ITERATION
#define K_QUANTS_PER_ITERATION 2
#else
static_assert(K_QUANTS_PER_ITERATION == 1 || K_QUANTS_PER_ITERATION == 2, "K_QUANTS_PER_ITERATION must be 1 or 2");
#endif

#include <algorithm>
#include <assert.h>
#include <atomic>
#include <climits>
#include <condition_variable>
#include <cstddef>
#include <cstdint>
#include <limits>
#include <mutex>
#include <queue>
#include <stdint.h>
#include <stdio.h>
#include <thread>
#include <unordered_map>
#include <unordered_set>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
#include <nvtx3/nvToolsExt.h>

#include "ggml.h"
#include "ggml-cuda.h"
#include "ggml-cuda-kern.h"
#include "ggml-cuda-quant.h"

#if defined(_MSC_VER)
#pragma warning(disable: 4244 4267) // possible loss of data
#endif

static_assert(sizeof(half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s (%s:%d): %s\n", err_,                  \
                __func__, __FILE__, __LINE__, hipGetErrorString(err_));                \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#if CUDART_VERSION >= 12000
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "\ncuBLAS error %d at %s (%s:%d): %s\n", err_,              \
                __func__, __FILE__, __LINE__, cublasGetStatusString(err_));             \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#else
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);  \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#endif // CUDART_VERSION >= 12000

#define UNUSED(x) (void)(x)

typedef void (*ggml_cuda_op_t)(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t cudaStream_main);

struct cuda_pool_buffer {
    void * ptr;
    size_t size;
};

static std::unordered_map<hipStream_t, std::vector<cuda_pool_buffer>> g_cuda_stream_pools;
static size_t g_cuda_pool_size = 0;

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size, hipStream_t stream) {
    std::vector<cuda_pool_buffer>& pool = g_cuda_stream_pools[stream];

    // find existing
    for (size_t i = 0; i < pool.size(); ++i) {
        cuda_pool_buffer& b = pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            pool.erase(pool.begin() + i);
            return ptr;
        }
    }

    // allocate new
    void * ptr;
    CUDA_CHECK(hipMalloc(&ptr, size));
    *actual_size = size;

    g_cuda_pool_size += size;

    //fprintf(stderr, "cuda pool size: %.2f MB (allocating now: %.2f MB)\n", g_cuda_pool_size / 1024.0 / 1024.0, size / 1024.0 / 1024.0);

    return ptr;
}

static void ggml_cuda_pool_free(void * ptr, size_t size, hipStream_t stream) {
    std::vector<cuda_pool_buffer>& pool = g_cuda_stream_pools[stream];

    pool.push_back({ ptr, size });
}

static void ggml_cuda_pool_free_all() {
    for (auto& p : g_cuda_stream_pools) {
        for (auto& b : p.second) {
            if (b.ptr != nullptr) {
                CUDA_CHECK(hipFree(b.ptr));
            }
        }
    }
    g_cuda_stream_pools.clear();
}

template<typename src_t>
static void quantize_row_q8_1_cuda(const src_t * x, void * vy, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
    quantize_q8_1<<<num_blocks, CUDA_QUANTIZE_BLOCK_SIZE, 0, stream>>>(x, vy, k);
}

template<typename dst_t>
static void dequantize_row_q4_0_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<dst_t, QK4_0, QR4_0, dequantize_q4_0<dst_t>><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

template<typename dst_t>
static void dequantize_row_q4_1_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<dst_t, QK4_1, QR4_1, dequantize_q4_1<dst_t>><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

template<typename dst_t>
static void dequantize_row_q5_0_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<dst_t, QK5_0, QR5_0, dequantize_q5_0<dst_t>><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

template<typename dst_t>
static void dequantize_row_q5_1_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<dst_t, QK5_1, QR5_1, dequantize_q5_1<dst_t>><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

template<typename dst_t>
static void dequantize_row_q8_0_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<dst_t, QK8_0, QR8_0, dequantize_q8_0<dst_t>><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

/*
static void dequantize_row_q2_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q2_K<<<nb, 64, 0, stream>>>(vx, y);
}

static void dequantize_row_q3_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q3_K<<<nb, 64, 0, stream>>>(vx, y);
}

template<typename dst_t>
static void dequantize_row_q4_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q4_K<<<nb, 32, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q5_K<<<nb, 64, 0, stream>>>(vx, y);
}

*/
template<typename dst_t>
static void dequantize_row_q6_K_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q6_K<<<nb, 64, 0, stream>>>(vx, y);
}

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q4_0_cuda(const void * vx, const src1_t * y, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<src1_t, dst_t, QK4_0, QR4_0, dequantize_q4_0<dst_t>>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q4_1_cuda(const void * vx, const src1_t * y, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<src1_t, dst_t, QK4_1, QR4_1, dequantize_q4_1<dst_t>>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q5_0_cuda(const void * vx, const src1_t * y, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<src1_t, dst_t, QK5_0, QR5_0, dequantize_q5_0<dst_t>>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q5_1_cuda(const void * vx, const src1_t * y, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<src1_t, dst_t, QK5_1, QR5_1, dequantize_q5_1<dst_t>>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q8_0_cuda(const void * vx, const src1_t * y, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<src1_t, dst_t, QK8_0, QR8_0, dequantize_q8_0<dst_t>>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}
/*
template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q2_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q2_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q3_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const dim3 block_dims(32, 1, 1);
    dequantize_mul_mat_vec_q3_k<<<nrows, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q4_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const dim3 block_dims(32, 1, 1);
    dequantize_mul_mat_vec_q4_k<<<nrows, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q5_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const dim3 block_dims(32, 1, 1);
    dequantize_mul_mat_vec_q5_k<<<nrows, block_dims, 0, stream>>>(vx, y, dst, ncols);
}
*/

template<typename src1_t, typename dst_t>
static void dequantize_mul_mat_vec_q6_K_cuda(const void * vx, const src1_t * y, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q6_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

template<typename src1_t, typename dst_t>
static void convert_mul_mat_vec_f16_cuda(const void * vx, const src1_t * y, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<src1_t, dst_t, 1, 1, convert_fp16<dst_t>><<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

template<typename dst_t>
static void mul_mat_vec_q4_0_q8_1_cuda(const void * vx, const void * vy, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    mul_mat_vec_q<dst_t, QK4_0, QI4_0, block_q4_0, vec_dot_q4_0_q8_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols, nrows);
}

template<typename dst_t>
static void mul_mat_vec_q4_1_q8_1_cuda(const void * vx, const void * vy, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    mul_mat_vec_q<dst_t, QK4_0, QI4_1, block_q4_1, vec_dot_q4_1_q8_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols, nrows);
}

template<typename dst_t>
static void mul_mat_vec_q5_0_q8_1_cuda(const void * vx, const void * vy, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    mul_mat_vec_q<dst_t, QK5_0, QI5_0, block_q5_0, vec_dot_q5_0_q8_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols, nrows);
}

template<typename dst_t>
static void mul_mat_vec_q5_1_q8_1_cuda(const void * vx, const void * vy, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    mul_mat_vec_q<dst_t, QK5_1, QI5_1, block_q5_1, vec_dot_q5_1_q8_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols, nrows);
}

template<typename dst_t>
static void mul_mat_vec_q8_0_q8_1_cuda(const void * vx, const void * vy, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    mul_mat_vec_q<dst_t, QK8_0, QI8_0, block_q8_0, vec_dot_q8_0_q8_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols, nrows);
}

template<typename dst_t>
static void convert_fp16_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<dst_t, 1, 1, convert_fp16<dst_t>><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

template<typename dst_t>
static to_t_cuda_t<dst_t> ggml_get_to_t_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        /*
        case GGML_TYPE_Q2_K:
            return dequantize_row_q2_K_cuda;
        case GGML_TYPE_Q3_K:
            return dequantize_row_q3_K_cuda;
        case GGML_TYPE_Q4_K:
            return dequantize_row_q4_K_cuda;
        case GGML_TYPE_Q5_K:
            return dequantize_row_q5_K_cuda;
        */
        case GGML_TYPE_Q6_K:
            return dequantize_row_q6_K_cuda;
        case GGML_TYPE_F16:
            return convert_fp16_cuda;
        default:
            return nullptr;
    }
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_mul_mat_p021_cuda(const src0_t * vx, const src1_t * y, dst_t * dst, const int ncols_x, const int nrows_x, const int nchannels_x, hipStream_t stream) {
    const dim3 block_nums(1, nrows_x, nchannels_x);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    k_mul_mat_p021<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols_x, nrows_x, nchannels_x);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_mul_mat_vec_nc_cuda(
    const src0_t * vx, const src1_t * y, dst_t * dst, const int ncols_x, const int nrows_x, const int row_stride_x,
    const int nchannels_x, const int channel_stride_x, hipStream_t stream) {

    const dim3 block_nums(1, nrows_x, nchannels_x);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    k_mul_mat_vec_nc<<<block_nums, block_dims, 0, stream>>>
        (vx, y, dst, ncols_x, nrows_x, row_stride_x, nchannels_x, channel_stride_x);
}

template<typename src_t, typename dst_t>
static void ggml_cpy_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int nb00, const int nb01, const int nb02,
    const int ne10, const int ne11, const int nb10, const int nb11, const int nb12, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    k_cpy<src_t, dst_t><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, nb00, nb01, nb02, ne10, ne11, nb10, nb11, nb12);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void add_cuda(const src0_t * x, const src1_t * y, dst_t * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_ADD_BLOCK_SIZE - 1) / CUDA_ADD_BLOCK_SIZE;
    k_add<<<num_blocks, CUDA_ADD_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void mul_cuda(const src0_t * x, const src1_t * y, dst_t * dst, const int kx, const int ky, hipStream_t stream) {
    const int num_blocks = (kx + CUDA_MUL_BLOCK_SIZE - 1) / CUDA_MUL_BLOCK_SIZE;
    k_mul<<<num_blocks, CUDA_MUL_BLOCK_SIZE, 0, stream>>>(x, y, dst, kx, ky);
}

template<typename src0_t, typename dst_t>
static void silu_cuda(const src0_t * x, dst_t * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    k_silu<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template<typename src0_t, typename dst_t>
static void rms_norm_cuda(const src0_t * x, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % WARP_SIZE == 0);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    k_rms_norm<<<nrows, block_dims, 0, stream>>>(x, dst, ncols);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void scale_cuda(const src0_t * x, dst_t * dst, const src1_t * scale, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SCALE_BLOCK_SIZE - 1) / CUDA_SCALE_BLOCK_SIZE;
    k_scale<<<num_blocks, CUDA_SCALE_BLOCK_SIZE, 0, stream>>>(x, dst, scale, k);
}

template<typename src0_t, typename dst_t>
static void rope_cuda(const src0_t * x, dst_t * dst, const int ncols, const int nrows, const float p, const float theta_scale, hipStream_t stream) {
    GGML_ASSERT(nrows % 2 == 0);
    const dim3 block_dims(2*CUDA_ROPE_BLOCK_SIZE, 1, 1);
    const int num_blocks_x = (ncols + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(num_blocks_x, nrows, 1);
    k_rope<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols, p, theta_scale);
}

template<typename src0_t, typename dst_t>
static void diag_mask_inf_cuda(const src0_t * x, dst_t * dst, const int ncols_x, const int nrows_x, const int rows_per_channel, const int n_past, hipStream_t stream) {
    const dim3 block_dims(CUDA_DIAG_MASK_INF_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ncols_x + CUDA_DIAG_MASK_INF_BLOCK_SIZE - 1) / CUDA_DIAG_MASK_INF_BLOCK_SIZE;
    const dim3 block_nums(block_num_x, nrows_x, 1);
    k_diag_mask_inf<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols_x, rows_per_channel, n_past);
}

template<typename src0_t, typename dst_t>
static void soft_max_cuda(const src0_t * x, dst_t * dst, const int ncols, const int nrows, hipStream_t stream) {
    // TODO: implement fast numerically stable version for small ncols
    //if (ncols >= 1024) {
        int num_blocks = nrows;
        if (ncols % 2 == 0) {
            k_soft_max<src0_t, dst_t, 2 , 1024>
                <<<num_blocks, 1024, 0, stream>>>(x, dst, nrows, ncols);
        }
        else {
            k_soft_max<src0_t, dst_t, 1, 1024>
                <<<num_blocks, 1024, 0, stream>>>(x, dst, nrows, ncols);
        }
    //}
    //else {
    //    const dim3 block_dims(WARP_SIZE, 1, 1);
    //    const dim3 block_nums(1, nrows, 1);
    //    k_soft_max_orig<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols);
    //}
}

template<typename dst_t, int qk, int qr, dequantize_kernel_t<dst_t> dq>
static void get_rows_cuda(const void * x, const int * y, dst_t * dst, const int nrows, const int ncols, hipStream_t stream) {
    const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
    const int block_num = (ncols/2 + CUDA_GET_ROWS_BLOCK_SIZE - 1) / CUDA_GET_ROWS_BLOCK_SIZE;
    const dim3 block_nums(block_num, nrows, 1);
    k_get_rows<dst_t, qk, qr, dq><<<block_nums, block_dims, 0, stream>>>(x, y, dst, ncols);
}

// TODO: move to context
static hipblasHandle_t g_cublas_handle = nullptr;
static hipStream_t g_cudaStream_main = nullptr;
static hipEvent_t g_cudaEvent_main = nullptr;
static hipStream_t g_cudaStreams[GGML_CUDA_MAX_SUBSTREAMS] = { };
static hipEvent_t g_cudaEvents[GGML_CUDA_MAX_SUBSTREAMS] = { };
#define GGML_CUDA_MAX_DEVICES 16
static int g_compute_capabilities[GGML_CUDA_MAX_DEVICES];

static void ggml_init_cublas() {
    static bool initialized = false;

    if (!initialized) {
        int device_count;
        CUDA_CHECK(hipGetDeviceCount(&device_count));
        int64_t total_vram = 0;
        fprintf(stderr, "%s: found %d CUDA devices:\n", __func__, device_count);
        for (int id = 0; id < device_count; ++id) {
            hipDeviceProp_t prop;
            CUDA_CHECK(hipGetDeviceProperties(&prop, id));
            fprintf(stderr, "  Device %d: %s (%.0f GB)\n", id, prop.name, prop.totalGlobalMem / 1024.0 / 1024.0 / 1024.0);
            total_vram += prop.totalGlobalMem;
            g_compute_capabilities[id] = 100*prop.major + 10*prop.minor;
        }

        // create main stream and event
        CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStream_main, hipStreamNonBlocking));
        CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvent_main, hipEventDisableTiming));

        // create secondary streams and events
        for (int i = 0; i < GGML_CUDA_MAX_SUBSTREAMS; ++i) {
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams[i], hipStreamNonBlocking));
            CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvents[i], hipEventDisableTiming));
        }

        // create cublas handle
        CUBLAS_CHECK(hipblasCreate(&g_cublas_handle));
        //CUBLAS_CHECK(hipblasSetMathMode(g_cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH));

        // configure logging to stdout
        //CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));

        initialized = true;
    }
}

void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        // The allocation error can be bypassed. A null ptr will assigned out of this function.
        // This can fixed the OOM error in WSL.
        hipGetLastError();
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_add(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    const int64_t ne0 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    add_cuda((src0_t *)src0_d, (src1_t *) src1_d, (dst_t *) dst_d, ne0*i01_diff, stream);
    CUDA_CHECK(hipGetLastError());

    UNUSED(src1);
    UNUSED(dst);
    UNUSED(i02);
    UNUSED(i1);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_mul(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    for (int64_t i01 = i01_low; i01 < i01_high; i01++) {
        const int64_t i11 = i1*ne11 + i01%ne11; // broadcast src1 across src0

        src0_t * src0_d_i01 = (src0_t *) src0_d + i01*ne00;
        src1_t * src1_d_i01 = (src1_t *) src1_d + i11*ne10;
        dst_t * dst_d_i01 = (dst_t *) dst_d + i01*ne00;

        // compute
        mul_cuda(src0_d_i01, src1_d_i01, dst_d_i01, ne00, ne10, stream);
        CUDA_CHECK(hipGetLastError());
    }

    UNUSED(dst);
    UNUSED(i02);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_silu(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    silu_cuda((src0_t *)src0_d, (dst_t *)dst_d, ne00*i01_diff, stream);
    CUDA_CHECK(hipGetLastError());

    UNUSED(src1);
    UNUSED(src1_d);
    UNUSED(dst);
    UNUSED(i02);
    UNUSED(i1);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_rms_norm(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    rms_norm_cuda((src0_t *)src0_d, (dst_t *)dst_d, ne00, i01_diff, stream);
    CUDA_CHECK(hipGetLastError());

    UNUSED(src1);
    UNUSED(src1_d);
    UNUSED(dst);
    UNUSED(i02);
    UNUSED(i1);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_dequantize_mul_mat_vec(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t nrows = i01_high - i01_low;

#ifdef GGML_CUDA_FORCE_DMMV
    const bool use_mul_mat_vec_q = false;
#else
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    const bool mul_mat_vec_q_implemented = src0->type == GGML_TYPE_Q4_0 ||
        src0->type == GGML_TYPE_Q4_1 ||
        src0->type == GGML_TYPE_Q5_0 ||
        src0->type == GGML_TYPE_Q5_1 ||
        src0->type == GGML_TYPE_Q8_0;

    // The integer intrinsics used in mul_mat_vec_q are available with compute capability 6.
    // However, they have bad performance with Pascal cards.
    // Therefore, in a multi GPU setting decide at runtime which GPUs should use mul_mat_vec_q.
    const bool use_mul_mat_vec_q = g_compute_capabilities[id] >= 700 && mul_mat_vec_q_implemented;
#endif

    if (use_mul_mat_vec_q) {
        size_t as;
        void * src1_q8_1 = ggml_cuda_pool_malloc(ne00*sizeof(block_q8_1)/QK8_1, &as, stream);
        quantize_row_q8_1_cuda((src1_t *)src1_d, src1_q8_1, ne00, stream);

        switch (src0->type) {
            case GGML_TYPE_Q4_0:
                mul_mat_vec_q4_0_q8_1_cuda(src0_d, src1_q8_1, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_Q4_1:
                mul_mat_vec_q4_1_q8_1_cuda(src0_d, src1_q8_1, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_Q5_0:
                mul_mat_vec_q5_0_q8_1_cuda(src0_d, src1_q8_1, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_Q5_1:
                mul_mat_vec_q5_1_q8_1_cuda(src0_d, src1_q8_1, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_Q8_0:
                mul_mat_vec_q8_0_q8_1_cuda(src0_d, src1_q8_1, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            default:
                GGML_ASSERT(false);
                break;
        }

        ggml_cuda_pool_free(src1_q8_1, as, stream);
    }
    else {
        switch (src0->type) {
            case GGML_TYPE_Q4_0:
                dequantize_mul_mat_vec_q4_0_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_Q4_1:
                dequantize_mul_mat_vec_q4_1_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_Q5_0:
                dequantize_mul_mat_vec_q5_0_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_Q5_1:
                dequantize_mul_mat_vec_q5_1_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_Q8_0:
                dequantize_mul_mat_vec_q8_0_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            /*
            case GGML_TYPE_Q2_K:
                dequantize_mul_mat_vec_q2_K_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, cudaStream_main);
                break;
            case GGML_TYPE_Q3_K:
                dequantize_mul_mat_vec_q3_K_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, cudaStream_main);
                break;
            case GGML_TYPE_Q4_K:
                dequantize_mul_mat_vec_q4_K_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, cudaStream_main);
                break;
            case GGML_TYPE_Q5_K:
                dequantize_mul_mat_vec_q5_K_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, cudaStream_main);
                break;
            */
            case GGML_TYPE_Q6_K:
                dequantize_mul_mat_vec_q6_K_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            case GGML_TYPE_F16:
                convert_mul_mat_vec_f16_cuda(src0_d, (src1_t *)src1_d, (dst_t *)dst_d, ne00, nrows, stream);
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
    }
    CUDA_CHECK(hipGetLastError());

    UNUSED(src1);
    UNUSED(dst);
    UNUSED(i02);
    UNUSED(i1);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_rope(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {


    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    const int n_past = ((int32_t *) dst->params)[0];
    const int n_dims = ((int32_t *) dst->params)[1];
    const int mode   = ((int32_t *) dst->params)[2];
    //const int n_ctx  = ((int32_t *) dst->params)[3];
    GGML_ASSERT(mode == 0);

    const float theta_scale = powf(10000.0, -2.0f/n_dims);
    const float p = ((mode & 1) == 0 ? n_past + i02 : i02);

    // compute
    rope_cuda((src0_t *)src0_d, (dst_t *)dst_d, ne00, i01_diff, p, theta_scale, stream);
    CUDA_CHECK(hipGetLastError());

    UNUSED(dst);
    UNUSED(src1);
    UNUSED(src1_d);
    UNUSED(i1);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_diag_mask_inf(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t i01_diff = i01_high - i01_low;

    const int n_past = ((int32_t *) dst->params)[0];

    // compute
    diag_mask_inf_cuda((src0_t *)src0_d, (dst_t *)dst_d, ne00, i01_diff, ne01, n_past, stream);
    CUDA_CHECK(hipGetLastError());

    UNUSED(dst);
    UNUSED(src1);
    UNUSED(src1_d);
    UNUSED(i02);
    UNUSED(i1);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_soft_max(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    soft_max_cuda((src0_t *)src0_d, (dst_t *)dst_d, ne00, i01_diff, stream);
    CUDA_CHECK(hipGetLastError());

    UNUSED(src1);
    UNUSED(src1_d);
    UNUSED(dst);
    UNUSED(i02);
    UNUSED(i1);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_scale(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    //const src1_t scale = ((src1_t *) src1->data)[0];

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    scale_cuda<src0_t, src1_t>((src0_t *)src0_d, (dst_t *)dst_d, (src1_t *)src1_d, ne00*i01_diff, stream);
    CUDA_CHECK(hipGetLastError());

    UNUSED(src1);
    UNUSED(src1_d);
    UNUSED(dst);
    UNUSED(i02);
    UNUSED(i1);
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_get_rows(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    GGML_ASSERT(src1->type == GGML_TYPE_I32);
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(ggml_is_contiguous(dst));

    const int ncols = src0->ne[0];
    const int nrows = ggml_nelements(src1);

    switch (src0->type) {
        case GGML_TYPE_F16:
            get_rows_cuda<dst_t, 1, 1, convert_fp16<dst_t>>(src0_d, (int *) src1_d, (dst_t *)dst_d, nrows, ncols, stream);
            break;
        case GGML_TYPE_F32:
            get_rows_cuda<dst_t, 1, 1, convert_fp32<dst_t>>(src0_d, (int *) src1_d, (dst_t *)dst_d, nrows, ncols, stream);
            break;
        case GGML_TYPE_Q4_0:
            get_rows_cuda<dst_t, QK4_0, QR4_0, dequantize_q4_0<dst_t>>(src0_d, (int *) src1_d, (dst_t *)dst_d, nrows, ncols, stream);
            break;
        case GGML_TYPE_Q4_1:
            get_rows_cuda<dst_t, QK4_1, QR4_1, dequantize_q4_1<dst_t>>(src0_d, (int *) src1_d, (dst_t *)dst_d, nrows, ncols, stream);
            break;
        case GGML_TYPE_Q5_0:
            get_rows_cuda<dst_t, QK5_0, QR5_0, dequantize_q5_0<dst_t>>(src0_d, (int *) src1_d, (dst_t *)dst_d, nrows, ncols, stream);
            break;
        case GGML_TYPE_Q5_1:
            get_rows_cuda<dst_t, QK5_1, QR5_1, dequantize_q5_1<dst_t>>(src0_d, (int *) src1_d, (dst_t *)dst_d, nrows, ncols, stream);
            break;
        case GGML_TYPE_Q8_0:
            get_rows_cuda<dst_t, QK8_0, QR8_0, dequantize_q8_0<dst_t>>(src0_d, (int *) src1_d, (dst_t *)dst_d, nrows, ncols, stream);
            break;

        default:
            GGML_ASSERT(false);
            break;
    }
    CUDA_CHECK(hipGetLastError());

    UNUSED(i02);
    UNUSED(i01_low);
    UNUSED(i01_high);
    UNUSED(i1);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

struct ggml_cuda_buffer {
    const char * name;

    void   * data;
    size_t   size;
    void   * device;
};

struct ggml_cuda_context {
    std::vector<ggml_cuda_buffer> buffers;
};

ggml_cuda_context * ggml_cuda_init() {
    ggml_init_cublas();

    ggml_cuda_context * ctx = new ggml_cuda_context;

    return ctx;
}

void ggml_cuda_free(ggml_cuda_context * ctx) {
    for (size_t n = 0; n < ctx->buffers.size(); ++n) {
        if (ctx->buffers[n].device != nullptr) {
            CUDA_CHECK(hipFree(ctx->buffers[n].device));
        }
    }

    // this will free the global memory pool for all contexts
    ggml_cuda_pool_free_all();

    delete ctx;
}

static void * ggml_cuda_get_buffer(ggml_cuda_context * ctx, ggml_tensor * t) {
    return t->data;

    UNUSED(ctx);
}

static hipError_t ggml_cuda_cpy_tensor_2d(ggml_cuda_context * ctx,
    void * dst, ggml_tensor * src, int64_t i3, int64_t i2, int64_t i1_low, int64_t i1_high, hipStream_t stream) {

    hipMemcpyKind kind = hipMemcpyDeviceToDevice;
    const char * src_ptr = (const char *) ggml_cuda_get_buffer(ctx, src);
    char * dst_ptr = (char *) dst;

    const int64_t ne0 = src->ne[0];
    const int64_t nb0 = src->nb[0];
    const int64_t nb1 = src->nb[1];
    const int64_t nb2 = src->nb[2];
    const int64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const int64_t ts = ggml_type_size(type);
    const int64_t bs = ggml_blck_size(type);
    int64_t i1_diff = i1_high - i1_low;

    GGML_ASSERT(i1_low == 0);
    const char * x = src_ptr + i1_low*nb1 + i2*nb2 + i3*nb3;
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst_ptr, x, i1_diff*nb1, kind, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst_ptr, ts*ne0/bs, x, nb1, ts*ne0/bs, i1_diff, kind, stream);
    } else {
        for (int64_t i1 = 0; i1 < i1_diff; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) (dst_ptr + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, kind, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

static const ggml_type GGML_TYPE_NONE = GGML_TYPE_COUNT;

struct ggml_cuda_op_dispatch_t {
    ggml_cuda_op_t d[GGML_TYPE_COUNT][GGML_TYPE_COUNT+1][GGML_TYPE_COUNT] = { nullptr };
};

template<template <typename src0_t, typename src1_t, typename dst_t> class Op>
static ggml_cuda_op_dispatch_t gen_op_dispatch_table() {
    ggml_cuda_op_dispatch_t dispatch;

    dispatch.d[GGML_TYPE_F16][GGML_TYPE_NONE][GGML_TYPE_F16] = &Op<half, half, half>::op;
    //dispatch.d[GGML_TYPE_F16][GGML_TYPE_NONE][GGML_TYPE_F32] = &Op<half, half, float>::op;
    dispatch.d[GGML_TYPE_F16][GGML_TYPE_F16][GGML_TYPE_F16] = &Op<half, half, half>::op;
    dispatch.d[GGML_TYPE_F16][GGML_TYPE_F16][GGML_TYPE_F32] = &Op<half, half, float>::op;
    dispatch.d[GGML_TYPE_F16][GGML_TYPE_F32][GGML_TYPE_F16] = &Op<half, float, half>::op;
    dispatch.d[GGML_TYPE_F16][GGML_TYPE_F32][GGML_TYPE_F32] = &Op<half, float, float>::op;
    //dispatch.d[GGML_TYPE_F32][GGML_TYPE_NONE][GGML_TYPE_F16] = &Op<float, float, half>::op;
    dispatch.d[GGML_TYPE_F32][GGML_TYPE_NONE][GGML_TYPE_F32] = &Op<float, float, float>::op;
    //dispatch.d[GGML_TYPE_F32][GGML_TYPE_F16][GGML_TYPE_F16] = &Op<float, half, half>::op;
    dispatch.d[GGML_TYPE_F32][GGML_TYPE_F16][GGML_TYPE_F32] = &Op<float, half, float>::op;
    //dispatch.d[GGML_TYPE_F32][GGML_TYPE_F32][GGML_TYPE_F16] = &Op<float, float, half>::op;
    dispatch.d[GGML_TYPE_F32][GGML_TYPE_F32][GGML_TYPE_F32] = &Op<float, float, float>::op;

    return dispatch;
}

template<template <typename src0_t, typename src1_t, typename dst_t> class Op>
static ggml_cuda_op_t get_op_fn(ggml_type t0, ggml_type t1, ggml_type t2) {
    static const ggml_cuda_op_dispatch_t dispatch = gen_op_dispatch_table<Op>();

    if (dispatch.d[t0][t1][t2] == nullptr) {
        fprintf(stderr, "Unsupported type combination: %s %s %s\n",
                ggml_type_name(t0), ggml_type_name(t1), ggml_type_name(t2));
    }

    GGML_ASSERT(dispatch.d[t0][t1][t2] && "Unsupported type combination");
    return dispatch.d[t0][t1][t2];
}

template<template <typename src0_t, typename src1_t, typename dst_t> class Op>
static void ggml_cuda_op(ggml_cuda_context * ctx,
                    ggml_tensor * src0,
                    ggml_tensor * src1,
                    ggml_tensor * dst,
                    hipStream_t main_stream,
                    bool flatten_rows) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];
    const int64_t nrows0 = ggml_nrows(src0);

    const bool use_src1 = src1 != nullptr;
    const int64_t ne10 = use_src1 ? src1->ne[0] : 1;
    const int64_t ne11 = use_src1 ? src1->ne[1] : 1;
    const int64_t ne12 = use_src1 ? src1->ne[2] : 1;
    const int64_t ne13 = use_src1 ? src1->ne[3] : 1;

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    ggml_type t0 = src0->type;
    ggml_type t1 = use_src1 ? src1->type : GGML_TYPE_NONE;
    ggml_type t2 = dst->type;
    // HACK
    // get rows
    if (t1 == GGML_TYPE_I32) {
        t1 = t2;
    }
    // mul mat
    if (ggml_is_quantized(t0)) {
        t0 = t1;
    }

    ggml_cuda_op_t op = get_op_fn<Op>(t0, t1, t2);

    //const int nb2  = dst->nb[2];
    //const int nb3  = dst->nb[3];

    // strides for iteration over dims 3 and 2
    const int64_t num_iters = flatten_rows ? 1 : ne02 * ne03;
    const int64_t stride_mod = flatten_rows ? ne02 * ne03 : 1;
    const int64_t src0_stride = ne00 * ne01 * stride_mod;
    const int64_t src1_stride = ne10 * ne11 * stride_mod;
    const int64_t dst_stride = ne0 * ne1 * stride_mod;

    const size_t src0_ts = ggml_type_size(src0->type);
    const size_t src0_bs = ggml_blck_size(src0->type);
    const size_t src1_ts = use_src1 ? ggml_type_size(src1->type) : 0;
    const size_t src1_bs = use_src1 ? ggml_blck_size(src1->type) : 1;
    const size_t dst_ts = ggml_type_size(dst->type);
    const size_t dst_bs = ggml_blck_size(dst->type);

    const bool src0_is_contiguous = ggml_is_contiguous(src0);
    const bool src1_is_contiguous = use_src1 ? ggml_is_contiguous(src1) : true;

    void * src0_d = src0 ? ggml_cuda_get_buffer(ctx, src0) : nullptr;
    void * src1_d = src1 ? ggml_cuda_get_buffer(ctx, src1) : nullptr;
    void * dst_d  = dst  ? ggml_cuda_get_buffer(ctx, dst)  : nullptr;

    int64_t row_low = 0;
    int64_t row_high = nrows0;
    int64_t row_diff = row_high - row_low;

    size_t src0_as = 0;
    size_t src1_as = 0;
    if (!src0_is_contiguous) {
        src0_d = (float *) ggml_cuda_pool_malloc(row_diff*ne00 * src0_ts/src0_bs, &src0_as, main_stream);
    }

    if (!src1_is_contiguous) {
        src1_d = (float *) ggml_cuda_pool_malloc(num_iters*src1_stride * src1_ts/src1_bs, &src1_as, main_stream);
    }

    const int64_t i03_max = flatten_rows ? 1 : ne03;
    const int64_t i02_max = flatten_rows ? 1 : ne02;
    const int64_t rows_per_iter = flatten_rows ? nrows0 : ne01;
    const int64_t num_ops = i03_max * i02_max;

    if (num_ops > 1 && GGML_CUDA_MAX_SUBSTREAMS > 1) {
        // record an event on the stream to synchronize the sub-streams
        CUDA_CHECK(hipEventRecord(g_cudaEvent_main, main_stream));
    }

    for (int64_t i03 = 0; i03 < i03_max; i03++) {
        const int64_t i13 = i03 % ne13;
        for (int64_t i02 = 0; i02 < i02_max; i02++) {
            const int64_t i12 = i02 % ne12;

            const int64_t i0 = i03*ne02 + i02;
            const int64_t i0_offset_low = row_low/rows_per_iter;
            //const int64_t i0_offset_high = row_high/rows_per_iter;

            int64_t i01_low = 0;
            int64_t i01_high = rows_per_iter;

            const int64_t i01_diff = i01_high - i01_low;
            if (i01_diff == 0) {
                continue;
            }
            const int64_t i11 = i13*ne12 + i12;

            hipStream_t op_stream;
            if (num_ops > 1 && GGML_CUDA_MAX_SUBSTREAMS > 1) {
                op_stream = g_cudaStreams[i0 % GGML_CUDA_MAX_SUBSTREAMS];
                // wait for the main stream to finish, but only the first time per sub-stream
                if (i0 < GGML_CUDA_MAX_SUBSTREAMS) {
                    CUDA_CHECK(hipStreamWaitEvent(op_stream, g_cudaEvent_main, 0));
                }
            } else {
                op_stream = main_stream;
            }
            // TODO: use different streams, record event, wait for all events on main stream at the end

            // for split tensors the data begins at i0 == i0_offset_low
            void * src0_d_i = (char *) src0_d + (i0 - i0_offset_low)*src0_stride*src0_ts/src0_bs;
            void * src1_d_i = (char *) src1_d + i11*src1_stride*src1_ts/src1_bs;
            void * dst_d_i  = (char *) dst_d + (i0 - i0_offset_low)*dst_stride*dst_ts/dst_bs;

            // copy src0, src1 to device if necessary
            // CUDA_CHECK(hipEventRecord(cudaEvent_memcpy_src1, cudaStream_memcpy_src1));
            if (!src0_is_contiguous) {
                CUDA_CHECK(ggml_cuda_cpy_tensor_2d(ctx, src0_d_i, src0, i03, i02, i01_low, i01_high, op_stream));
            }
            if (!src1_is_contiguous) {
                CUDA_CHECK(ggml_cuda_cpy_tensor_2d(ctx, src1_d_i, src1, i03, i02, 0, ne11, op_stream));
            }

            op(src0, src1, dst,
                src0_d_i, src1_d_i, dst_d_i,
                i02, i01_low, i01_high, i11,
                op_stream);

            if (num_ops > 1 && GGML_CUDA_MAX_SUBSTREAMS > 1) {
                // record an event on the stream to synchronize with the main stream
                // only wait for the event if it is the last operation in this stream
                if (i0 >= (num_ops - GGML_CUDA_MAX_SUBSTREAMS)) {
                    CUDA_CHECK(hipEventRecord(g_cudaEvents[i0 % GGML_CUDA_MAX_SUBSTREAMS], op_stream));
                }
            }
        }
    }

    if (num_ops > 1 && GGML_CUDA_MAX_SUBSTREAMS > 1) {
        // wait for all events on the main stream
        for (int64_t i0 = 0; i0 < std::min((int)num_ops, GGML_CUDA_MAX_SUBSTREAMS); i0++) {
            // wait on the main stream for the event
            CUDA_CHECK(hipStreamWaitEvent(main_stream, g_cudaEvents[i0], 0));
        }
    }

    if (src1_as > 0) {
        ggml_cuda_pool_free(src1_d, src1_as, main_stream);
    }
    if (src0_as > 0) {
        ggml_cuda_pool_free(src0_d, src0_as, main_stream);
    }
}

static void ggml_cuda_cpy(ggml_cuda_context * ctx, ggml_tensor * src0, ggml_tensor * src1, ggml_tensor * dst, hipStream_t stream) {
    const int64_t ne = ggml_nelements(src0);
    GGML_ASSERT(ne == ggml_nelements(src1));

    GGML_ASSERT(ggml_nbytes(src0) <= INT_MAX);
    GGML_ASSERT(ggml_nbytes(src1) <= INT_MAX);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    GGML_ASSERT(src0->ne[3] == 1);

    const int64_t nb00 = src0->nb[0];
    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];
    GGML_ASSERT(src1->ne[3] == 1);

    const int64_t nb10 = src1->nb[0];
    const int64_t nb11 = src1->nb[1];
    const int64_t nb12 = src1->nb[2];

    hipStream_t cudaStream_main = stream;

    void * d_src0 = src0 ? ggml_cuda_get_buffer(ctx, src0) : nullptr;
    void * d_src1 = src1 ? ggml_cuda_get_buffer(ctx, src1) : nullptr;

    if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_cuda<float, float>((char *) d_src0, (char *) d_src1, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_cuda<half, half>((char *) d_src0, (char *) d_src1, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_cuda<half, float>((char *) d_src0, (char *) d_src1, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_cuda<float, half>((char *) d_src0, (char *) d_src1, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else if (src0->type == GGML_TYPE_I32 && src1->type == GGML_TYPE_I32) {
        ggml_cpy_cuda<int32_t, int32_t>((char *) d_src0, (char *) d_src1, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else {
        GGML_ASSERT(false);
    }
    CUDA_CHECK(hipGetLastError());

    UNUSED(dst);
}

static void ggml_cuda_mul_mat_vec_p021(ggml_cuda_context * ctx, ggml_tensor * src0, ggml_tensor * src1, ggml_tensor * dst, hipStream_t stream) {
    GGML_ASSERT(ggml_is_permuted(src0) && ggml_is_permuted(src1));
    GGML_ASSERT(src0->nb[0] <= src0->nb[1] && src0->nb[2] <= src0->nb[3]); // 0213 permutation
    GGML_ASSERT(src1->nb[0] <= src1->nb[1] && src1->nb[2] <= src1->nb[3]); // 0213 permutation

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    hipStream_t cudaStream_main = stream;

    void * src0_d = src0 ? ggml_cuda_get_buffer(ctx, src0) : nullptr;
    void * src1_d = src1 ? ggml_cuda_get_buffer(ctx, src1) : nullptr;
    void * dst_d  = dst  ? ggml_cuda_get_buffer(ctx, dst)  : nullptr;

    if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F16) {
        ggml_mul_mat_p021_cuda<half, half, half>((half *)src0_d, (half *)src1_d, (half *)dst_d, ne00, ne01, ne02, cudaStream_main);
    }
    else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32) {
        ggml_mul_mat_p021_cuda<half, float, float>((half *)src0_d, (float *)src1_d, (float *)dst_d, ne00, ne01, ne02, cudaStream_main);
    }
    else {
        GGML_ASSERT(false);
    }
}

static void ggml_cuda_mul_mat_vec_nc(ggml_cuda_context * ctx, ggml_tensor * src0, ggml_tensor * src1, ggml_tensor * dst, hipStream_t stream) {
    GGML_ASSERT(!ggml_is_contiguous(src0) && ggml_is_contiguous(src1));
    GGML_ASSERT(!ggml_is_permuted(src0));

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];

    hipStream_t cudaStream_main = stream;

    void * src0_d = src0 ? ggml_cuda_get_buffer(ctx, src0) : nullptr;
    void * src1_d = src1 ? ggml_cuda_get_buffer(ctx, src1) : nullptr;
    void * dst_d  = dst  ? ggml_cuda_get_buffer(ctx, dst)  : nullptr;

    const int row_stride_x = nb01 / ggml_type_size(src0->type);
    const int channel_stride_x = nb02 / ggml_type_size(src0->type);

    if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F16) {
        ggml_mul_mat_vec_nc_cuda<half, half, half>((half *)src0_d, (half *)src1_d, (half *)dst_d, ne00, ne01, row_stride_x, ne02, channel_stride_x, cudaStream_main);
    }
    else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32) {
        ggml_mul_mat_vec_nc_cuda<half, float, float>((half *)src0_d, (float *)src1_d, (float *)dst_d, ne00, ne01, row_stride_x, ne02, channel_stride_x, cudaStream_main);
    }
    else {
        GGML_ASSERT(false);
    }
}

static hipDataType ggml_to_cuda_type(ggml_type t) {
    switch (t) {
        case GGML_TYPE_F16: return HIP_R_16F;
        case GGML_TYPE_F32: return HIP_R_32F;
        default: puts(ggml_type_name(t)); GGML_ASSERT(false);
    }
}

template<typename src0_t, typename src1_t, typename dst_t>
static void ggml_cuda_op_mul_mat_cublas(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    void * src0_d, void * src1_d, void * dst_d,
    int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int64_t ne0 = dst->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // the main device has a larger memory buffer to hold the results from all GPUs
    // ldc == nrows of the matrix that cuBLAS writes into
    const int ldc = ne0; //dst->backend == GGML_BACKEND_GPU && id == g_main_device ? ne0 : i01_diff;

    ggml_type ts0 = src0->type;
    ggml_type ts1 = src1->type;
    ggml_type td = dst->type;

    size_t src0_as = 0;
    hipblasComputeType_t compute_type;

    if (ts0 == GGML_TYPE_F16 && ts1 == GGML_TYPE_F16 && td == GGML_TYPE_F16) {
        compute_type = HIPBLAS_COMPUTE_16F;
    }
    else if (ts0 == GGML_TYPE_F32 && ts1 == GGML_TYPE_F32 && td == GGML_TYPE_F32) {
        compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
    }
    else if (ts1 == GGML_TYPE_F32 && td == GGML_TYPE_F32) {
        compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;

        int ne = i01_diff * ne00;
        void * src0_f32 = ggml_cuda_pool_malloc(ne * sizeof(float), &src0_as, stream);

        const to_t_cuda_t<float> to_fp32_cuda = ggml_get_to_t_cuda<float>(src0->type);
        GGML_ASSERT(to_fp32_cuda);
        //printf("converting %s from %s\n", src0->name, ggml_type_name(src0->type));
        to_fp32_cuda(src0_d, (float *)src0_f32, ne, stream);
        CUDA_CHECK(hipGetLastError());
        src0_d = src0_f32;
        ts0 = GGML_TYPE_F32;
    }
    else if (ts1 == GGML_TYPE_F16) {
        if (td == GGML_TYPE_F16) {
            compute_type = HIPBLAS_COMPUTE_16F;
        }
        else if (td == GGML_TYPE_F32) {
            compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
        }
        else {
            GGML_ASSERT(false);
        }

        int ne = i01_diff * ne00;
        void * src0_f16 = ggml_cuda_pool_malloc(ne * sizeof(half), &src0_as, stream);

        const to_t_cuda_t<half> to_fp16_cuda = ggml_get_to_t_cuda<half>(src0->type);
        GGML_ASSERT(to_fp16_cuda);

        to_fp16_cuda(src0_d, (half *)src0_f16, ne, stream);
        CUDA_CHECK(hipGetLastError());
        src0_d = src0_f16;
        ts0 = GGML_TYPE_F16;
    }
    else {
        fprintf(stderr, "cuBLAS: unsupported types: %s * %s -> %s\n",
            ggml_type_name(ts0), ggml_type_name(ts1), ggml_type_name(td));
        GGML_ASSERT(false);
    }

    half alpha_f16 = 1.0f;
    half beta_f16 = 0.0f;
    float alpha_f32 = 1.0f;
    float beta_f32 = 0.0f;
    const void * alpha;
    const void * beta;

    switch (compute_type) {
        case HIPBLAS_COMPUTE_16F:
            alpha = &alpha_f16; beta = &beta_f16;
            break;
        case HIPBLAS_COMPUTE_32F_FAST_TF32:
        case HIPBLAS_COMPUTE_32F:
            alpha = &alpha_f32; beta = &beta_f32;
            break;
        default:
            GGML_ASSERT(false);
            break;
    }

    CUBLAS_CHECK(hipblasSetStream(g_cublas_handle, stream));
    CUBLAS_CHECK(
        hipblasGemmEx(g_cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                i01_diff, ne11, ne10,
                alpha, src0_d, ggml_to_cuda_type(ts0), ne00,
                       src1_d, ggml_to_cuda_type(ts1), ne10,
                beta,  dst_d,  ggml_to_cuda_type(td), ldc,
                compute_type,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    if (src0_as) {
        ggml_cuda_pool_free(src0_d, src0_as, stream);
    }

    UNUSED(i02);
    UNUSED(i1);
}

#define DEFINE_GGML_CUDA_OP_S(op_name)                                                              \
    template<typename src0_t, typename src1_t, typename dst_t>                                      \
    struct ggml_cuda_op_ ## op_name ## _s {                                                         \
        static void op(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,       \
                       void * src0_d, void * src1_d, void * dst_d,                                  \
                       int64_t i02, int64_t i01_low, int64_t i01_high, int i1,                      \
                       hipStream_t stream) {                                                       \
            ggml_cuda_op_ ## op_name<src0_t, src1_t, dst_t>(src0, src1, dst,                        \
                src0_d, src1_d, dst_d,                                                              \
                i02, i01_low, i01_high, i1,                                                         \
                stream);                                                                            \
        }                                                                                           \
    }

DEFINE_GGML_CUDA_OP_S(add);
DEFINE_GGML_CUDA_OP_S(mul);
DEFINE_GGML_CUDA_OP_S(scale);
DEFINE_GGML_CUDA_OP_S(mul_mat_cublas);
DEFINE_GGML_CUDA_OP_S(dequantize_mul_mat_vec);
DEFINE_GGML_CUDA_OP_S(silu);
DEFINE_GGML_CUDA_OP_S(soft_max);
DEFINE_GGML_CUDA_OP_S(diag_mask_inf);
DEFINE_GGML_CUDA_OP_S(rms_norm);
DEFINE_GGML_CUDA_OP_S(rope);
DEFINE_GGML_CUDA_OP_S(get_rows);

#undef DEFINE_GGML_CUDA_OP_S

static void ggml_cuda_mul_mat(ggml_cuda_context * ctx, ggml_tensor * src0, ggml_tensor * src1, ggml_tensor * dst, hipStream_t stream) {
    if (ggml_is_permuted(src0) && ggml_is_permuted(src1) && src1->ne[1] == 1) {
        ggml_cuda_mul_mat_vec_p021(ctx, src0, src1, dst, stream);
    } else if (!ggml_is_contiguous(src0) && ggml_is_contiguous(src1) && src1->ne[1] == 1) {
        ggml_cuda_mul_mat_vec_nc(ctx, src0, src1, dst, stream);
    } else {
        if (src1->ne[1] == 1 && src0->ne[0] % GGML_CUDA_DMMV_X == 0 && src0->ne[1] % GGML_CUDA_DMMV_Y == 0) {
            ggml_cuda_op<ggml_cuda_op_dequantize_mul_mat_vec_s>(ctx, src0, src1, dst, stream, false);
        } else {
            ggml_cuda_op<ggml_cuda_op_mul_mat_cublas_s>(ctx, src0, src1, dst, stream, false);
        }
    }
}

static void ggml_cuda_exec_node(ggml_cuda_context * ctx, ggml_tensor * node, hipStream_t stream) {
    ggml_tensor * src0 = node->src0;
    ggml_tensor * src1 = node->src1;
    ggml_tensor * dst  = node;

#if 0
    fprintf(stdout, "%s: %s %s %s %s (%s, %s, %s) %d\n",
                dst->name,
                ggml_op_name(dst->op),
                src0 ? ggml_type_name(src0->type) : "null",
                src1 ? ggml_type_name(src1->type) : "null",
                dst  ? ggml_type_name(dst->type)  : "null",
                src0 ? ggml_get_name(src0) : "null",
                src1 ? ggml_get_name(src1) : "null",
                dst  ? ggml_get_name(dst)  : "null",
                src1 ? ggml_is_contiguous(src1) : -1
            );
#endif
    switch ((int)dst->op) {
        case GGML_OP_RESHAPE:
        case GGML_OP_VIEW:
        case GGML_OP_TRANSPOSE:
        case GGML_OP_PERMUTE:
        case GGML_OP_NONE:
            {
                // noop
            } break;
        case GGML_OP_ADD:
            {
                ggml_cuda_op<ggml_cuda_op_add_s>(ctx, src0, src1, dst, stream, true);
            } break;
        case GGML_OP_MUL:
            {
                ggml_cuda_op<ggml_cuda_op_mul_s>(ctx, src0, src1, dst, stream, false); // TODO ggml_cuda_op needs modification for flatten
            } break;
        case GGML_OP_SCALE:
            {
                ggml_cuda_op<ggml_cuda_op_scale_s>(ctx, src0, src1, dst, stream, true);
            } break;
        case GGML_OP_SILU:
            {
                ggml_cuda_op<ggml_cuda_op_silu_s>(ctx, src0, src1, dst, stream, true);
            } break;
        case GGML_OP_SOFT_MAX:
            {
                ggml_cuda_op<ggml_cuda_op_soft_max_s>(ctx, src0, src1, dst, stream, true);
            } break;
        case GGML_OP_DIAG_MASK_INF:
            {
                ggml_cuda_op<ggml_cuda_op_diag_mask_inf_s>(ctx, src0, src1, dst, stream, true);
            } break;
        case GGML_OP_MUL_MAT:
            {
                ggml_cuda_mul_mat(ctx, src0, src1, dst, stream);
            } break;
        case GGML_OP_GET_ROWS:
            {
                ggml_cuda_op<ggml_cuda_op_get_rows_s>(ctx, src0, src1, dst, stream, true);
            } break;
        case GGML_OP_RMS_NORM:
            {
                ggml_cuda_op<ggml_cuda_op_rms_norm_s>(ctx, src0, src1, dst, stream, true);
            } break;
        case GGML_OP_ROPE:
            {
                ggml_cuda_op<ggml_cuda_op_rope_s>(ctx, src0, src1, dst, stream, false); // FIXME flatten changes results
            } break;
        case GGML_OP_CPY:
            {
                ggml_cuda_cpy(ctx, src0, src1, dst, stream);
            } break;
        default:
            fprintf(stderr, "%s: op = %8s not implemented\n", __func__, ggml_op_name(dst->op));
            GGML_ASSERT(false);
    }
}

static const int GGML_MAX_PARENTS = 2 + GGML_MAX_OPT;

static bool ggml_is_noop(ggml_tensor * t) {
    return t->op == GGML_OP_RESHAPE || t->op == GGML_OP_VIEW || t->op == GGML_OP_TRANSPOSE ||
           t->op == GGML_OP_PERMUTE || t->op == GGML_OP_NONE;
}

// TODO: reduce number of streams and events
static void ggml_cuda_graph_exec_parallel(ggml_cuda_context * ctx, ggml_cgraph * gf, hipStream_t mainStream) {
    // record an event for the nodes to add a dependency on
    hipEvent_t mainEvent = g_cudaEvent_main;

    CUDA_CHECK(hipEventRecord(mainEvent, mainStream));

    // TODO: move to context and free
    static std::vector<hipStream_t> free_streams;
    static std::vector<hipEvent_t> free_events;

    // TODO: preserve the order to allow reusing pool allocations
    free_streams.insert(free_streams.begin(), mainStream);

    std::unordered_set<hipStream_t> node_streams;
    std::vector<hipEvent_t> node_events;
    std::unordered_map<ggml_tensor *, hipEvent_t> event_map;
    std::unordered_map<ggml_tensor *, hipStream_t> stream_map;

    for (int i = 0; i < gf->n_nodes; ++i) {
        ggml_tensor * node = gf->nodes[i];
        const bool is_noop = ggml_is_noop(node);

        // build a list of parents
        ggml_tensor * parents[GGML_MAX_PARENTS] = { node->src0, node->src1 };
        for (int j = 0; j < GGML_MAX_OPT; j++) {
            parents[j + 2] = node->opt[j];
        }

        // assign an stream for the node
        hipStream_t stream = nullptr;

        // take a stream from a parent
        for (int j = 0; j < GGML_MAX_PARENTS; j++) {
            if (parents[j] && stream_map.count(parents[j]) && stream_map[parents[j]] != nullptr) {
                stream = stream_map[parents[j]];
                stream_map.erase(parents[j]);

                if (is_noop) {
                    // if this is a noop, we can use the parent's event
                    stream_map[node] = stream;
                    if (event_map.count(parents[j]) > 0) {
                        event_map[node] = event_map[parents[j]];
                    }
                }
                break;
            }
        }

        if (is_noop) {
            continue;
        }

        // otherwise, create a new stream
        if (!stream) {
            if (!free_streams.empty()) {
                stream = free_streams.back();
                free_streams.pop_back();
            }
            else {
                CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
            }
        }

        // wait on parent streams
        bool waited = false;
        for (int j = 0; j < GGML_MAX_PARENTS; j++) {
            if (parents[j] && event_map.count(parents[j]) > 0) {
                CUDA_CHECK(hipStreamWaitEvent(stream, event_map[parents[j]], 0));
                waited = true;
            }
        }

        // wait on the start event to introduce a dependency if no parents
        if (!waited) {
            CUDA_CHECK(hipStreamWaitEvent(stream, mainEvent, 0));
        }

        // execute the node
        ggml_cuda_exec_node(ctx, node, stream);

        // record an event for the node
        hipEvent_t event;
        if (!free_events.empty()) {
            event = free_events.back();
            free_events.pop_back();
        }
        else {
            CUDA_CHECK(hipEventCreateWithFlags(&event, hipEventDisableTiming));
        }

        CUDA_CHECK(hipEventRecord(event, stream));

        // save stream and event
        if (stream != mainStream) {
            node_streams.insert(stream);
        }
        node_events.push_back(event);
        event_map[node] = event;
        stream_map[node] = stream;
    }

    // wait for the group streams to finish
    for (auto & it : node_events) {
        CUDA_CHECK(hipStreamWaitEvent(mainStream, it, 0));
    }

    //printf("used %d events and %d streams\n", (int)node_events.size(), (int)node_streams.size());

    // save streams and events for reuse
    free_streams.insert(free_streams.end(), node_streams.begin(), node_streams.end());
    free_events.insert(free_events.end(), node_events.begin(), node_events.end());
}

static void ggml_cuda_synchronize(struct ggml_cuda_context * ctx) {
    CUDA_CHECK(hipStreamSynchronize(g_cudaStream_main));

    UNUSED(ctx);
}

static void ggml_cuda_cgraph_compute(ggml_cuda_context * ctx, ggml_cgraph * gf) {
    hipStream_t stream = g_cudaStream_main;

    if (GGML_CUDA_SEQ_COMPUTE) {
        for (int i = 0; i < gf->n_nodes; ++i) {
            ggml_cuda_exec_node(ctx, gf->nodes[i], stream);
        }
    }
    else {
        ggml_cuda_graph_exec_parallel(ctx, gf, stream);
    }
}

// backend interface

struct ggml_backend_cuda_context {
    ggml_cuda_context * cuda_ctx = ggml_cuda_init();
};

static const char * ggml_backend_cuda_name(ggml_backend_context_t ctx) {
    return "CUDA";

    UNUSED(ctx);
}

static void ggml_backend_cuda_free_context(ggml_backend_context_t ctx) {
    ggml_backend_cuda_context * cuda_ctx = (ggml_backend_cuda_context *)ctx;
    ggml_cuda_free(cuda_ctx->cuda_ctx);
    delete cuda_ctx;
}

struct cuda_backend_buffer {
    void * data;
    size_t offset;
    size_t size;
};

static const size_t TENSOR_ALIGNMENT = 128;

static size_t aligned_offset(const void * buffer, size_t offset, size_t alignment) {
    assert(alignment && !(alignment & (alignment - 1))); // power of 2
    size_t align = (alignment - (((uintptr_t)buffer + offset) % alignment)) % alignment;
    return offset + align;
}

static ggml_backend_buffer_t ggml_backend_cuda_alloc_buffer(ggml_backend_context_t ctx, size_t size) {
    cuda_backend_buffer * buffer = new cuda_backend_buffer;

    CUDA_CHECK(hipMalloc(&buffer->data, size));
    buffer->offset = 0; // hipMalloc returns aligned pointers
    buffer->size = size;

    return buffer;

    UNUSED(ctx);
}

static void ggml_backend_cuda_free_buffer(ggml_backend_context_t ctx, ggml_backend_buffer_t buffer) {
    cuda_backend_buffer * cuda_buffer = (cuda_backend_buffer *)buffer;
    CUDA_CHECK(hipFree(cuda_buffer->data));
    delete cuda_buffer;

    UNUSED(ctx);
}

static void ggml_backend_cuda_reset_buffer(ggml_backend_context_t ctx, ggml_backend_buffer_t buffer) {
    cuda_backend_buffer * cuda_buffer = (cuda_backend_buffer *)buffer;
    cuda_buffer->offset = 0;

    UNUSED(ctx);
}

static void ggml_backend_cuda_alloc_tensor(ggml_backend_context_t ctx, ggml_backend_buffer_t buffer, ggml_tensor * tensor) {
    cuda_backend_buffer * cuda_buffer = (cuda_backend_buffer *)buffer;

    if (cuda_buffer->offset + ggml_nbytes(tensor) > cuda_buffer->size) {
        fprintf(stderr, "%s: not enough space in the CUDA buffer (needed %zu, available %zu)\n",
                __func__, ggml_nbytes(tensor), cuda_buffer->size - cuda_buffer->offset);
        GGML_ASSERT(false);
    }

    tensor->data = (char*)cuda_buffer->data + cuda_buffer->offset;
    cuda_buffer->offset = aligned_offset(cuda_buffer->data, cuda_buffer->offset + ggml_nbytes(tensor), TENSOR_ALIGNMENT);

    UNUSED(ctx);
}

static void ggml_backend_cuda_set_tensor_async(ggml_backend_context_t ctx, ggml_tensor * tensor, const void * data, size_t offset, size_t size) {
    GGML_ASSERT(offset + size <= ggml_nbytes(tensor) && "tensor write out of bounds");
    GGML_ASSERT(tensor->data != NULL && "tensor not allocated");

    //ggml_backend_cuda_context * cuda_ctx = (ggml_backend_cuda_context *)ctx;

    CUDA_CHECK(hipMemcpyAsync((char*)tensor->data + offset, data, size, hipMemcpyHostToDevice, g_cudaStream_main));

    UNUSED(ctx);
}

static void ggml_backend_cuda_get_tensor_async(ggml_backend_context_t ctx, const ggml_tensor * tensor, void * data, size_t offset, size_t size) {
    GGML_ASSERT(offset + size <= ggml_nbytes(tensor) && "tensor read out of bounds");
    GGML_ASSERT(tensor->data != NULL && "tensor not allocated");

    //ggml_backend_cuda_context * cuda_ctx = (ggml_backend_cuda_context *)ctx;

    CUDA_CHECK(hipMemcpyAsync(data, (const char*)tensor->data + offset, size, hipMemcpyDeviceToHost, g_cudaStream_main));

    UNUSED(ctx);
}

static void ggml_backend_cuda_synchronize(ggml_backend_context_t ctx) {
    ggml_backend_cuda_context * cuda_ctx = (ggml_backend_cuda_context *)ctx;
    ggml_cuda_synchronize(cuda_ctx->cuda_ctx);
}

static ggml_graph_plan_t ggml_backend_cuda_graph_plan_create(ggml_backend_context_t ctx, ggml_cgraph * cgraph) {
    GGML_ASSERT(false);

    return nullptr;

    UNUSED(ctx);
    UNUSED(cgraph);
}

static void ggml_backend_cuda_graph_plan_free(ggml_backend_context_t ctx, ggml_graph_plan_t plan) {
    GGML_ASSERT(false);

    UNUSED(ctx);
    UNUSED(plan);
}

static void ggml_backend_cuda_graph_plan_compute(ggml_backend_context_t ctx, ggml_graph_plan_t plan) {
    GGML_ASSERT(false);

    UNUSED(ctx);
    UNUSED(plan);
}

static void ggml_backend_cuda_graph_compute(ggml_backend_context_t ctx, ggml_cgraph * cgraph) {
    ggml_backend_cuda_context * cuda_ctx = (ggml_backend_cuda_context *)ctx;
    ggml_cuda_cgraph_compute(cuda_ctx->cuda_ctx, cgraph);
}

static ggml_backend_interface cuda_backend_interface = {
    /* .get_name            = */ ggml_backend_cuda_name,
    /* .free_context        = */ ggml_backend_cuda_free_context,
    /* .alloc_buffer        = */ ggml_backend_cuda_alloc_buffer,
    /* .free_buffer         = */ ggml_backend_cuda_free_buffer,
    /* .reset_buffer        = */ ggml_backend_cuda_reset_buffer,
    /* .alloc_tensor        = */ ggml_backend_cuda_alloc_tensor,
    /* .set_tensor_async    = */ ggml_backend_cuda_set_tensor_async,
    /* .get_tensor_async    = */ ggml_backend_cuda_get_tensor_async,
    /* .synchronize         = */ ggml_backend_cuda_synchronize,
    /* .cpy_tensor_from     = */ nullptr,
    /* .cpy_tensor_to       = */ nullptr,
    /* .graph_plan_create   = */ ggml_backend_cuda_graph_plan_create,
    /* .graph_plan_free     = */ ggml_backend_cuda_graph_plan_free,
    /* .graph_plan_compute  = */ ggml_backend_cuda_graph_plan_compute,
    /* .graph_compute       = */ ggml_backend_cuda_graph_compute
};

ggml_backend ggml_backend_cuda_init(void) {
    ggml_backend_cuda_context * ctx = new ggml_backend_cuda_context;

    ggml_backend cuda_backend = {
        /* .interface = */ &cuda_backend_interface,
        /* .context   = */ ctx
    };
    return cuda_backend;
}
